
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <ctime>
#include <string>
#include <cmath>
#include <algorithm>

using namespace std;

struct Result {
    vector<int> arr;
    int array_size;
    int key;
    double sequential_time;
    double parallel_time;
    double speedup;
    int search_index;
};

void printResult(const Result& result) {
    cout << "{\n";
    cout << "\"result\": {\n";
    cout << "\"arr_size\": " << result.array_size << ",\n";
    cout << "\"key\": " << result.key << ",\n";
    cout << "\"arr\": [";
    for (size_t i = 0; i < result.arr.size(); ++i) {
        cout << result.arr[i];
        if (i != result.arr.size() - 1)
            cout << ", ";
    }
    cout << "],\n";
    cout << "\"seq_time\": " << result.sequential_time << ",\n";
    cout << "\"par_time\": " << result.parallel_time << ",\n";
    if (!isnan(result.speedup)) {
        cout << "\"speedup\": " << result.speedup << ",\n";
    } else {
        cout << "\"speedup\": null,\n"; // Handle NaN speedup as null
    }
    cout << "\"search_index\": " << result.search_index << "\n";
    cout << "}\n";
    cout << "}\n";
}


int binarySearchSequential(const vector<int>& arr, int key) {
    int low = 0;
    int high = arr.size() - 1;

    while (low <= high) {
        int mid = low + (high - low) / 2;
        if (arr[mid] == key)
            return mid;
        else if (arr[mid] < key)
            low = mid + 1;
        else
            high = mid - 1;
    }

    return -1; // key not found
}

__global__ void binarySearchParallel(const int* arr, int key, int* result, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        if (arr[tid] == key) {
            *result = tid;
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        cerr << "Usage: " << argv[0] << " <array_size> <key> <array_element_1> <array_element_2> ...\n";
        return 1;
    }

    int size = stoi(argv[1]);
    int key = stoi(argv[2]);
    vector<int> arr(size);

    for (int i = 0; i < size; i++) {
        arr[i] = stoi(argv[3 + i]);
    }

    sort(arr.begin(), arr.end());

    int* d_arr, *d_result;
    hipMalloc((void**)&d_arr, size * sizeof(int));
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_arr, arr.data(), size * sizeof(int), hipMemcpyHostToDevice);

    // Timing for parallel binary search
    clock_t start_parallel = clock();
    binarySearchParallel<<<1, size>>>(d_arr, key, d_result, size);
    hipDeviceSynchronize();
    clock_t end_parallel = clock();
    double time_parallel = ((double)(end_parallel - start_parallel)) / CLOCKS_PER_SEC;

    int h_result;
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Timing for sequential binary search
    clock_t start_sequential = clock();
    int index_sequential = binarySearchSequential(arr, key);
    clock_t end_sequential = clock();
    double time_sequential = ((double)(end_sequential - start_sequential)) / CLOCKS_PER_SEC;

    // Calculate speedup
    double speedup = time_sequential / time_parallel;

    // Create a JSON-like object
    Result result;
    result.arr = arr;
    result.array_size = size;
    result.key = key;
    result.sequential_time = time_sequential;
    result.parallel_time = time_parallel;
    result.speedup = speedup;
    result.search_index = index_sequential != -1 ? index_sequential : h_result;

    printResult(result);

    // Free device memory
    hipFree(d_arr);
    hipFree(d_result);

    return 0;
}